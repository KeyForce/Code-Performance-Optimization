#include "hip/hip_runtime.h"
/*
 * CLion使用VS作为编译器 平台设置为AMD64
 * CMakeLists 添加find_package(CUDA) find_package(CUDA REQUIRED)
*/
#include <iostream>
#include <math.h>
#include "timer.h"
// 两个向量加法kernel，grid和block均为一维 只采用一个GPU
__global__
void add_one_cuda(int n, float *x, float *y)
{
    for (int i = 0; i < n; i++)
        y[i] = x[i] + y[i];
}

int main(void)
{
    int N = 1<<20;
    float *x, *y;

    // 申请device内存
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // 申请host内存
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // 执行kernel
    add_one_cuda<<<1, 1>>>(N, x, y);

    // 保证device和host数据同步，这样不用手动将device得到的结果拷贝到host
    hipDeviceSynchronize();

    // 计算误差
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // 释放device内存
    hipFree(x);
    hipFree(y);

    return 0;
}